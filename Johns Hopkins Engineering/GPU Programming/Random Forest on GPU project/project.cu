//////////////////////////////////////////////////////
// Paul Aoun										//
//													//
// Johns Hopkins University							//				
//													//
// 12/13/2020										//
//													//
// This is the main source code file.            	//
// It has the logic to setup, initialize, and       //
// generate the Random Forest using both the host   //
// and device for the implementation.               //
//////////////////////////////////////////////////////

#include <iostream>
#include <fstream>
#include <vector>
#include <string>
#include <algorithm>
#include <random>
#include <chrono>

#include "project.cuh"
#include "RandomForest.h"
#include "Node.h"

//Data file to generate the tree for the random forest
#define FILE_NAME "spy.us.short.csv"
//Comma delimited CSV file
//Split charater to read the data file. 44 is a "," comma.
#define SPLIT_CHAR 44

//Decision column in the data file based on which the Gini impurities will be calculated
#define DECISION_COLUMN 20 

#define START_COLUMN 10 //first column in the data file to be used in the Trees
#define MAX_RANDOM_NUMBERS 1000 //used to get the column numbers between MIN and MAX
#define MIN_COLUMN_NUMBER 10 // lowest column number to be included in the tree building
#define NUMBER_OF_TREES 5 //Number of trees per forest


// Flag are Pageable, Page-locked or Portable host memory [0, 1, or 2]
#define FLAG 0

//Generate a Tree for the Random Forest, using recursion and depth-first algorithm
Node* generateTree (int decisionColumn, std::vector<Node*>&nodes, 
    int numberOfNodes, int numberOfSelectedNodes, Node *rootNode) {

    if (DEBUG) 
    {
        std::cout << "generateTree: decisionColumn=" << decisionColumn << std::endl;
    }

    //Initialize the impurity with 2 (impossible) since the Gini values are between 0 and 1.
    rootNode->setGiniImpurity(2.0f);
    std::vector<std::vector<std::string>> records = rootNode->getRecords();

    //Find root node with the lowest Gini impurity
    int nodeCounter1{0};   
    for (Node *node : nodes) { 
        if (rootNode->getGiniImpurity() > node->getGiniImpurity()) {
            rootNode = node;
            if (DEBUG)
                std::cout << "generateTree: Temp root node for Tree is column " << rootNode->getColumnNumber() 
                    << " with Gini impurity " << rootNode->getGiniImpurity() << std::endl;
        }
    }

    //Prepare the record for the left side and right side of the binary tree
    std::vector<std::vector<std::string>> leftRecords; 
    std::vector<std::vector<std::string>> rightRecords; 

    //Remove column data from the rows
    for (std::vector<std::string> &row : records) {
        leftRecords.push_back(row);
        rightRecords.push_back(row);
    }

    std::cout << "generateTree: Final root node column number is " << rootNode->getColumnNumber() 
            << " with Gini impurity of " << rootNode->getGiniImpurity() << std::endl;
   
    
    //Recursive depth-first generation of the tree
    if (nodes.size() > numberOfNodes - numberOfSelectedNodes + 1) 
    {
        //Remove current node from the vector and reset Gini impurities for the nodes
        int nodeCounter{0};
        for (Node *tempNode : nodes) {
            if (tempNode->getColumnNumber() == rootNode->getColumnNumber()) 
            {
                nodes.erase(nodes.begin() + nodeCounter);
                if (DEBUG)
                    std::cout <<"generateTree: set node=" << rootNode->getColumnNumber() 
                        << " in Tree." <<std::endl; 
            }
            else
            {
                tempNode->setGiniImpurity(2.0f);
            }
            nodeCounter++;
        }
        
        if (DEBUG)
            for (Node *node: nodes) {
                std::cout << "generateTree: Node column number=" << node->getColumnNumber() << std::endl;
            }
        // Calculate Gini impurities for the selected columns and store in the nodes
        if (nodes.size() > numberOfNodes - numberOfSelectedNodes + 1) {

            //Calculate right-side Gini impurities for the selected columns and store in the nodes
            calculateColumnGini(decisionColumn, leftRecords, nodes);
            Node *leftNode = new Node(rootNode->getId() + 1);
            leftNode->setRecords(leftRecords);
            //Recursively call the generateTree to continue building it
            rootNode->setLeftNode(generateTree (decisionColumn, nodes, numberOfNodes, numberOfSelectedNodes, leftNode));

            //Calculate right-side Gini impurities for the selected columns and store in the nodes
            calculateColumnGini(decisionColumn, rightRecords, nodes);
            Node *rightNode = new Node (rootNode->getId() + 1);
            //Recursively call the generateTree to continue building it
            rightNode->setRecords(rightRecords);
            rootNode->setRightNode(generateTree (decisionColumn, nodes, numberOfNodes, numberOfSelectedNodes, rightNode));
        } 
    } else 
    {//Leaf node and end of the recursion
        std::cout << "generateTree: recursion completed and Tree generated." << std::endl;
    }
    
    return rootNode;
}

//Setup root node for the tree and call generateTree to recursively build the tree
void generateTreeControl (RandomForest randomForest, int numberOfTrees, int decisionColumn) {

    std::vector<std::vector<std::string>> bootStrapRecords;

    float *randomNumbers;
    int numberOfRows = randomForest.getNumberOfRows();
    int numberOfColumns = randomForest.getNumberOfColumns();

    //Allocate and initialize host arrays
    allocateArraysHost (numberOfRows, &randomNumbers, FLAG);
    randomInitializeArray(numberOfRows, &randomNumbers, numberOfColumns);

    //Get the bootstrapRecords from the Forest object
    bootStrapRecords = 
        randomForest.getBootStrapRecords (FILE_NAME, SPLIT_CHAR, numberOfRows, &randomNumbers);

 
    //Release the random numbers array
    freeArraysHost(&randomNumbers, FLAG);
    
    std::vector<Node*> nodes = randomForest.getAllNodes();

    //Get the columns to use in generating the tree
    selectColumns(MIN_COLUMN_NUMBER, numberOfColumns, DECISION_COLUMN, MAX_RANDOM_NUMBERS, 
        FLAG, nodes);

    int numberOfSelectedNodes{0};
    if (DEBUG)
        for (Node *node : nodes) 
        {
            std::cout << "generateRandomForest: Column " << node->getColumnNumber() 
                << " selected=" << node->getSelectedColumn() << std::endl;
                if (node->getSelectedColumn() == 1)
                    numberOfSelectedNodes++;
        }
    
    // Calculate Gini impurities for the selected columns and store in the nodes
    calculateColumnGini(DECISION_COLUMN, bootStrapRecords, nodes);
   
    if (DEBUG)
        for (Node *node : nodes) {
            std::cout << "Gini impurity for column " << node->getColumnNumber() 
                << " is " << node->getGiniImpurity() << std::endl;
        }

    std::chrono::high_resolution_clock::time_point t1 = std::chrono::high_resolution_clock::now();
    //Root node for the Tree
    Node *node = new Node(0);
    node->setRecords(bootStrapRecords);
    //Call generateTree with the root node for the Tree
    Node *rootNode = generateTree(DECISION_COLUMN, nodes, 
        numberOfColumns, numberOfSelectedNodes, node);
    std::chrono::high_resolution_clock::time_point t2 = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> time_span = std::chrono::duration_cast<std::chrono::duration<double>>(t2 - t1);	
    std::cout << "Tree generation completed in "<< time_span.count() << "ms." << std::endl;

}

//Generate the number of Trees for the Random Forest based on numberOfTrees
//Decision column is used to calculate the Gini impurities
void generateRandomForest (int numberOfTrees, int decisionColumn) 
{
    std::cout << std::fixed;

    //Create RandomForest object. The constructor will load the data in a vector of string
    RandomForest randomForest (FILE_NAME, SPLIT_CHAR);

    //Repeatedly generate a tree up to numberOfTrees
    std::chrono::high_resolution_clock::time_point t1 = std::chrono::high_resolution_clock::now();
    for (int i=0; i<numberOfTrees; i++) 
    {
        std::cout << "Tree number " << i << " generation started." << std::endl;
        generateTreeControl(randomForest, numberOfTrees, decisionColumn);
        std::cout << "Tree number " << i << " generation completed." << std::endl;
    }
    std::chrono::high_resolution_clock::time_point t2 = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> time_span = std::chrono::duration_cast<std::chrono::duration<double>>(t2 - t1);	
    std::cout << "Random Forest of " << numberOfTrees << " completed in "<< time_span.count() << "ms." << std::endl;

}

int main () {

    std::cout << "Welcome to Intro to GPU - Project." << std::endl;

    //Generate the Random Forst with the required number of Trees,
    // and based on the decision column
    generateRandomForest(NUMBER_OF_TREES, DECISION_COLUMN);

    /* Reset device after being done with all the functions */
	hipDeviceReset();    
    
    return 0;
}